#include "hip/hip_runtime.h"
#include "../../common/hip/hip_runtime_api.h"
#include "../../common/book.h"
#include <stdio.h>
#include <iostream>

__global__ void add(int a , int b, int* c) {
    *c = a + b;
}


int main(void){
    

    hipDeviceProp_t prop;

    int count;
    HANDLE_ERROR(hipGetDeviceCount(&count));

    for(int i = 0; i < count; i++){
        HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
        printf("    --- General information for device %d ---\n",i);
        printf("Name:  %s\n", prop.name );
        printf("Compute capability: %d.%d \n" , prop.major , prop.minor);
        printf("Clock rate:  %d\n" , prop.clockRate);
        printf("Device copy overlap:   ");
        if(prop.deviceOverlap)
            printf("Enabled\n");
        else
            printf("Disabled\n");

        printf("Kernel execition timeout:   ");
        if(prop.kernelExecTimeoutEnabled)
            printf("Enabled\n");
        else    
            printf("Disabled\n");

        printf("   --- Memory Information for device %d ---\n",i);
        printf("Total global memory: %ld\n", prop.totalGlobalMem/1000000);
        printf("Total constant memory: %ld\n", prop.totalConstMem/1000000);
        printf("Max mem pitch: %ld\n", prop.memPitch);
        printf("Texture Alignment: %ld\n", prop.textureAlignment);
        printf("   --- MP Information for device %d ---\n",i);
        printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
        printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
        printf("Registers per mp: %d\n", prop.regsPerBlock);
        printf("Threads in warp:  %d\n", prop.warpSize);
        printf("Max threads per block:  %d\n" , prop.maxThreadsPerBlock);
        printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0] ,prop.maxThreadsDim[1],prop.maxThreadsDim[2]);
        printf("Max grid dimensions: (%d, %d, %d)\n\n", prop.maxGridSize[0],prop.maxGridSize[1],prop.maxGridSize[2]);

    }

    return 0;
}